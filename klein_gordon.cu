#include <iostream>
#include <vector>
#include <cmath>
#include <fstream>
#include <string>

#include <hip/hip_runtime.h>


//simulation parameters
const int N_x = 500; //how many points there are on the 1D grid
const int N_steps = 500; //amount of time steps in the simulation
const int steps_per_data_write = 50; //amount of steps transpiring per each time the data is saved

const double domain_size = 10.0; //size of the physical domain
const double dx = domain_size / N_x; //distance between grid points on the physical domain (spatial step size)
const double dt = 0.9 * dx; //time step

//since we're working in natural units, c = h_bar = 1,
//so the only numerical value in the Klein-Gordon equation we need to worry about is mass
const double m = 5.0; // mass

//Gaussian parameters
const double A = 1.0; // amplitude of wave packet
const double x_0 = domain_size / 2.0; // initial position of the wave packet, initialized to center of domain
const double sigma = 0.5;  // Gaussian width
const double k = 3.0; // wave number


//error checking function
static void checkCudaError(hipError_t result, const char* msg)
{
    if (result != hipSuccess) {
        std::cerr << "CUDA error (" << msg << "): "
                  << hipGetErrorString(result) << std::endl;
        exit(EXIT_FAILURE);
    }
}

void set_initial_conditions(std::vector<double>& phi, std::vector<double>& prev_phi)
{
    for (int i = 0; i < N_x; i++)
    {
        double x_i = i * dx; //physical coordinate which corresponds to the i-th grid point
        phi[i] = A * std::exp(-1.0 * std::pow((x_i - x_0), 2) / (2.0 * sigma * sigma)) * std::cos(k * x_i); // initialization of each field point based on Gaussian wave packet
        prev_phi[i] = phi[i]; // each future time step requires knowledge of the previous and present time steps
    }
}

//CUDA kernel that calculates each time step
__global__ void time_step_kernel(const double* __restrict__ phi, const double* __restrict__ prev_phi, double* __restrict__ new_phi, double dx, double dt, double m, int N_x)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i >= 1 && i < (N_x - 1))
    {
        double laplacian = (phi[i - 1] - 2.0 * phi[i] + phi[i + 1]) / (dx * dx); //computes the approximate laplacian given by the finite difference method

        new_phi[i] = 2.0 * phi[i] - prev_phi[i] + (dt * dt) * (laplacian - (m * m) * phi[i]); //next field point calculation
    }
}


//CUDA kernel that applies periodic boundary conditions
__global__ void apply_periodic_bc_kernel(double* __restrict__ new_phi, int N_x)
{
    //since we want to enforce periodic boundary conditions it is necessary to make sure the edges of the field wrap around
    new_phi[0] = new_phi[N_x - 2];
    new_phi[N_x - 1] = new_phi[1];
}



void write_data(const std::vector<double>& phi, int step)
{
    std::ofstream file("klein_gordon_output_" + std::to_string(step) + ".dat");
    for (int i = 0; i < N_x; i++)
    {
        file << i * dx << " " << phi[i] << "\n";
    }
    file.close();
}


int main()
{
    //field arrays
    std::vector<double> h_phi(N_x, 0.0);
    std::vector<double> h_prev_phi(N_x, 0.0);
    std::vector<double> h_new_phi(N_x, 0.0);

    //sets initial conditions
    set_initial_conditions(h_phi, h_prev_phi);

    //pointer declaration
    double* d_phi       = nullptr;
    double* d_prev_phi  = nullptr;
    double* d_new_phi   = nullptr;


    //gpu memory allocation
    checkCudaError(hipMalloc((void**)&d_phi, N_x * sizeof(double)), "alloc d_phi");
    checkCudaError(hipMalloc((void**)&d_prev_phi, N_x * sizeof(double)), "alloc d_prev_phi");
    checkCudaError(hipMalloc((void**)&d_new_phi, N_x * sizeof(double)), "alloc d_new_phi");

    //copies data from the CPU to GPU
    checkCudaError(hipMemcpy(d_phi, h_phi.data(), N_x * sizeof(double), hipMemcpyHostToDevice), "cpy phi H->D");
    checkCudaError(hipMemcpy(d_prev_phi, h_prev_phi.data(), N_x * sizeof(double), hipMemcpyHostToDevice), "cpy prev_phi H->D");

    
    //kernel parameters
    const int blockSize = 128;
    const int gridSize  = (N_x + blockSize - 1)/blockSize;

    //time evolution loop
    for (int step = 0; step < N_steps; step++)
    {
        //launches time_step_kernel
        time_step_kernel<<<gridSize, blockSize>>>(d_phi, d_prev_phi, d_new_phi, dx, dt, m, N_x);
        checkCudaError(hipGetLastError(), "time_step_kernel");

        //launches periodic boundary conditions kernel
        //<<<1, 1>>> as arguments because only two values are being updated 
        apply_periodic_bc_kernel<<<1, 1>>>(d_new_phi, N_x);
        checkCudaError(hipGetLastError(), "apply_periodic_bc_kernel");

        //makes sure that everything is working
        checkCudaError(hipDeviceSynchronize(), "kernel sync");

        
        //variable swapping
        double* temp = d_prev_phi;
        d_prev_phi = d_phi;
        d_phi      = d_new_phi;
        d_new_phi  = temp;
    

        std::cout << "completed step " << step << "\n";
     
        if (step % steps_per_data_write == 0 || step == N_steps - 1)
        {
            checkCudaError(hipMemcpy(h_phi.data(), d_phi, N_x * sizeof(double), hipMemcpyDeviceToHost), "cpy phi D->H");

            write_data(h_phi, step);
            std::cout << "wrote data for step " << step << "\n";
        }
    }

    //cleans up device memory
    hipFree(d_phi);
    hipFree(d_prev_phi);
    hipFree(d_new_phi);

    return 0;
}
